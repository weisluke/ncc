#include "hip/hip_runtime.h"
/******************************************************************************

Please provide credit to Luke Weisenbach should this code be used.
Email: weisluke@alum.mit.edu

******************************************************************************/


#include "ncc.cuh"
#include "util.hpp"

#include <iostream>
#include <limits> //for std::numeric_limits
#include <string>


using dtype = double;
NCC<dtype> ncc;

/******************************************************************************
constants to be used
******************************************************************************/
constexpr int OPTS_SIZE = 2 * 9;
const std::string OPTS[OPTS_SIZE] =
{
	"-h", "--help",
	"-v", "--verbose",
	"-ip", "--infile_prefix",
	"-hl", "--half_length",
	"-px", "--pixels",
	"-os", "--over_sample",
	"-wm", "--write_maps",
	"-wh", "--write_histograms",
	"-o", "--outfile_prefix"
};

/******************************************************************************
default input option values
******************************************************************************/
bool verbose = false;



/******************************************************************************
Print the program usage help message

\param name -- name of the executable
******************************************************************************/
void display_usage(char* name)
{
	if (name)
	{
		std::cout << "Usage: " << name << " opt1 val1 opt2 val2 opt3 val3 ...\n";
	}
	else
	{
		std::cout << "Usage: programname opt1 val1 opt2 val2 opt3 val3 ...\n";
	}
	std::cout
		<< "                                                                               \n"
		<< "Options:\n"
		<< "  -h,--help               Show this help message.\n"
		<< "  -v,--verbose            Toggle verbose output. Takes no option value.\n"
		<< "  -ip,--infile_prefix     Specify the prefix to be used when reading in files.\n"
		<< "                          Default value: " << ncc.infile_prefix << "\n"
		<< "  -hl,--half_length       Specify the half-length of the square source plane\n"
		<< "                          region to find the number of caustic crossings in.\n"
		<< "                          Default value: " << ncc.half_length << "\n"
		<< "  -px,--pixels            Specify the number of pixels per side for the number\n"
		<< "                          of caustic crossings map. Default value: " << ncc.num_pixels << "\n"
		<< "  -os,--over_sample       Specify the power of 2 by which to oversample the\n"
		<< "                          final pixels. E.g., an input of 4 means the final\n"
		<< "                          pixel array will initially be oversampled by a value\n"
		<< "                          of 2^4 = 16 along both axes. This will require\n"
		<< "                          16*16 = 256 times more memory. Default value: " << ncc.over_sample << "\n"
		<< "  -wm,--write_maps        Specify whether to write number of caustic crossings\n"
		<< "                          maps (1) or not (0). Default value: " << ncc.write_maps << "\n"
		<< "  -wh,--write_histograms  Specify whether to write histograms (1) or not (0).\n"
		<< "                          Default value: " << ncc.write_histograms << "\n"
		<< "  -o,--outfile_prefix     Specify the prefix to be used in output file names.\n"
		<< "                          Default value: " << ncc.outfile_prefix << "\n";
}



int main(int argc, char* argv[])
{
	/******************************************************************************
	if help option has been input, display usage message
	******************************************************************************/
	if (cmd_option_exists(argv, argv + argc, "-h") || cmd_option_exists(argv, argv + argc, "--help"))
	{
		display_usage(argv[0]);
		return -1;
	}

	/******************************************************************************
	if there are input options, but not an even number (since all options take a
	parameter), display usage message and exit
	subtract 1 to take into account that first argument array value is program name
	account for possible verbose option, which is a toggle and takes no input
	******************************************************************************/
	if ((argc - 1) % 2 != 0 &&
		!(cmd_option_exists(argv, argv + argc, "-v") || cmd_option_exists(argv, argv + argc, "--verbose")))
	{
		std::cerr << "Error. Invalid input syntax.\n";
		display_usage(argv[0]);
		return -1;
	}

	/******************************************************************************
	check that all options given are valid. use step of 2 since all input options
	take parameters (assumed to be given immediately after the option). start at 1,
	since first array element, argv[0], is program name
	account for possible verbose option, which is a toggle and takes no input
	******************************************************************************/
	for (int i = 1; i < argc; i += 2)
	{
		if (argv[i] == std::string("-v") || argv[i] == std::string("--verbose"))
		{
			verbose = true;
			i--;
			continue;
		}
		if (!cmd_option_valid(OPTS, OPTS + OPTS_SIZE, argv[i]))
		{
			std::cerr << "Error. Invalid input syntax. Unknown option " << argv[i] << "\n";
			display_usage(argv[0]);
			return -1;
		}
	}


	/******************************************************************************
	BEGIN read in options and values, checking correctness and exiting if necessary
	******************************************************************************/

	char* cmdinput = nullptr;

	for (int i = 1; i < argc; i += 2)
	{
		/******************************************************************************
		account for possible verbose option, which is a toggle and takes no input
		******************************************************************************/
		if (argv[i] == std::string("-v") || argv[i] == std::string("--verbose"))
		{
			i--;
			continue;
		}

		cmdinput = cmd_option_value(argv, argv + argc, std::string(argv[i]));

		if (argv[i] == std::string("-ip") || argv[i] == std::string("--infile_prefix"))
		{
			set_param("infile_prefix", ncc.infile_prefix, cmdinput, verbose);
		}
		else if (argv[i] == std::string("-hl") || argv[i] == std::string("--half_length"))
		{
			try
			{
				set_param("half_length", ncc.half_length, std::stod(cmdinput), verbose);
				if (ncc.half_length < std::numeric_limits<dtype>::min())
				{
					std::cerr << "Error. Invalid half_length input. half_length must be >= " << std::numeric_limits<dtype>::min() << "\n";
					return -1;
				}
			}
			catch (...)
			{
				std::cerr << "Error. Invalid half_length input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-px") || argv[i] == std::string("--pixels"))
		{
			try
			{
				set_param("num_pixels", ncc.num_pixels, std::stoi(cmdinput), verbose);
				if (ncc.num_pixels < 1)
				{
					std::cerr << "Error. Invalid num_pixels input. num_pixels must be an integer > 0\n";
					return -1;
				}
			}
			catch (...)
			{
				std::cerr << "Error. Invalid num_pixels input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-os") || argv[i] == std::string("--over_sample"))
		{
			try
			{
				set_param("over_sample", ncc.over_sample, std::stoi(cmdinput), verbose);
				if (ncc.over_sample < 0)
				{
					std::cerr << "Error. Invalid over_sample input. over_sample must be an integer >= 0\n";
					return -1;
				}
			}
			catch (...)
			{
				std::cerr << "Error. Invalid over_sample input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-wm") || argv[i] == std::string("--write_maps"))
		{
			try
			{
				set_param("write_maps", ncc.write_maps, std::stoi(cmdinput), verbose);
				if (ncc.write_maps != 0 && ncc.write_maps != 1)
				{
					std::cerr << "Error. Invalid write_maps input. write_maps must be 1 (true) or 0 (false).\n";
					return -1;
				}
			}
			catch (...)
			{
				std::cerr << "Error. Invalid write_maps input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-wh") || argv[i] == std::string("--write_histograms"))
		{
			try
			{
				set_param("write_histograms", ncc.write_histograms, std::stoi(cmdinput), verbose);
				if (ncc.write_histograms != 0 && ncc.write_histograms != 1)
				{
					std::cerr << "Error. Invalid write_histograms input. write_histograms must be 1 (true) or 0 (false).\n";
					return -1;
				}
			}
			catch (...)
			{
				std::cerr << "Error. Invalid write_histograms input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-o") || argv[i] == std::string("--outfile_prefix"))
		{
			set_param("outfile_prefix", ncc.outfile_prefix, cmdinput, verbose);
		}
	}

	std::cout << "\n";

	/******************************************************************************
	END read in options and values, checking correctness and exiting if necessary
	******************************************************************************/


	/******************************************************************************
	check that a CUDA capable device is present
	******************************************************************************/
	int n_devices = 0;

	hipGetDeviceCount(&n_devices);
	if (cuda_error("hipGetDeviceCount", false, __FILE__, __LINE__)) return -1;

	if (verbose)
	{
		std::cout << "Available CUDA capable devices:\n\n";

		for (int i = 0; i < n_devices; i++)
		{
			hipDeviceProp_t prop;
			hipGetDeviceProperties(&prop, i);
			if (cuda_error("hipGetDeviceProperties", false, __FILE__, __LINE__)) return -1;

			show_device_info(i, prop);
			std::cout << "\n";
		}
	}

	if (n_devices > 1)
	{
		std::cout << "More than one CUDA capable device detected. Defaulting to first device.\n\n";
	}
	hipSetDevice(0);
	if (cuda_error("hipSetDevice", false, __FILE__, __LINE__)) return -1;


	/******************************************************************************
	run and save files
	******************************************************************************/
	if (!ncc.run(verbose)) return -1;
	if (!ncc.save(verbose)) return -1;


	std::cout << "Done.\n";

	hipDeviceReset();
	if (cuda_error("hipDeviceReset", false, __FILE__, __LINE__)) return -1;

	return 0;
}

