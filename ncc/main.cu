#include "hip/hip_runtime.h"
/*****************************************************************

Please provide credit to Luke Weisenbach should this code be used.
Email: weisluke@alum.mit.edu

*****************************************************************/


#include "complex.cuh"
#include "ncc_microlensing.cuh"
#include "ncc_read_write_files.cuh"
#include "util.hpp"

#include <algorithm>
#include <iostream>
#include <fstream>
#include <chrono>
#include <limits>
#include <new>
#include <string>


using dtype = double;

/*constants to be used*/
constexpr int OPTS_SIZE = 2 * 10;
const std::string OPTS[OPTS_SIZE] =
{
	"-h", "--help",
	"-ip", "--infile_prefix",
	"-it", "--infile_type",
	"-hl", "--half_length",
	"-px", "--pixels",
	"-os", "--over_sample",
	"-wm", "--write_maps",
	"-wh", "--write_histograms",
	"-ot", "--outfile_type",
	"-o", "--outfile_prefix"
};


/*default input option values*/
std::string infile_prefix = "./";
std::string infile_type = ".bin";
dtype half_length = static_cast<dtype>(5);
int num_pixels = 1000;
int over_sample = 2;
int write_maps = 1;
int write_histograms = 1;
std::string outfile_prefix = "./";
std::string outfile_type = ".bin";

/*default variable values*/
const std::string caustics_parameter_file = "ccf_parameter_info.txt";
const std::string caustics_file = "ccf_caustics";

int num_rows = 0;
int num_cols = 0;



/************************************
Print the program usage help message

\param name -- name of the executable
************************************/
void display_usage(char* name)
{
	if (name)
	{
		std::cout << "Usage: " << name << " opt1 val1 opt2 val2 opt3 val3 ...\n";
	}
	else
	{
		std::cout << "Usage: programname opt1 val1 opt2 val2 opt3 val3 ...\n";
	}
	std::cout
		<< "                                                                               \n"
		<< "Options:\n"
		<< "  -h,--help               Show this help message\n"
		<< "  -ip,--infile_prefix     Specify the prefix to be used when reading in files.\n"
		<< "                          Default value: " << infile_prefix << "\n"
		<< "  -it,--infile_type       Specify the type of input file to be used when\n"
		<< "                          reading in files. Default value: " << infile_type << "\n"
		<< "  -hl,--half_length       Specify the half-length of the square source plane\n"
		<< "                          region to find the number of caustic crossings in.\n"
		<< "                          Default value: " << half_length << "\n"
		<< "  -px,--pixels            Specify the number of pixels per side for the number\n"
		<< "                          of caustic crossings map. Default value: " << num_pixels << "\n"
		<< "  -os,--over_sample       Specify the power of 2 by which to oversample the\n"
		<< "                          final pixels. E.g., an input of 4 means the final\n"
		<< "                          pixel array will initially be oversampled by a value\n"
		<< "                          of 2^4 = 16 along both axes. This will require\n"
		<< "                          16*16 = 256 times more memory. Default value: " << over_sample << "\n"
		<< "  -wp,--write_maps        Specify whether to write the number of caustic\n"
		<< "                          crossings map. Default value: " << write_maps << "\n"
		<< "  -wh,--write_histograms  Specify whether to write the histogram (1)\n"
		<< "                          or not (0). Default value: " << write_histograms << "\n"
		<< "  -ot,--outfile_type      Specify the type of file to be output. Valid options\n"
		<< "                          are binary (.bin) or text (.txt). Default value: " << outfile_type << "\n"
		<< "  -o,--outfile            Specify the prefix to be used in output file names.\n"
		<< "                          Default value: " << outfile_prefix << "\n"
		<< "                          Lines of output files are whitespace delimited.\n"
		<< "                          Filenames are:\n"
		<< "                            ncc_parameter_info  various parameter values used\n"
		<< "                                                  in calculations\n"
		<< "                            ncc_ncc_numpixels   each line contains a number of\n"
		<< "                                                  caustic crossings and the\n"
		<< "                                                  number of pixels with that\n"
		<< "                                                  many caustic crossings\n"
		<< "                            ncc_ncc             the first item is num_pixels\n"
		<< "                                                  and the second item is\n"
		<< "                                                  num_pixels followed by the\n"
		<< "                                                  number of caustic crossings\n"
		<< "                                                  at the center of each pixel\n";
}



int main(int argc, char* argv[])
{
	/*if help option has been input, display usage message*/
	if (cmd_option_exists(argv, argv + argc, std::string("-h")) || cmd_option_exists(argv, argv + argc, std::string("--help")))
	{
		display_usage(argv[0]);
		return -1;
	}

	/*if there are input options, but not an even number (since all options
	take a parameter), display usage message and exit
	subtract 1 to take into account that first argument array value is program name*/
	if ((argc - 1) % 2 != 0)
	{
		std::cerr << "Error. Invalid input syntax.\n";
		display_usage(argv[0]);
		return -1;
	}

	/*check that all options given are valid. use step of 2 since all input
	options take parameters (assumed to be given immediately after the option)
	start at 1, since first array element, argv[0], is program name*/
	for (int i = 1; i < argc; i += 2)
	{
		if (!cmd_option_valid(OPTS, OPTS + OPTS_SIZE, argv[i]))
		{
			std::cerr << "Error. Invalid input syntax. Unknown option " << argv[i] << "\n";
			display_usage(argv[0]);
			return -1;
		}
	}

	/******************************************************************************
	BEGIN read in options and values, checking correctness and exiting if necessary
	******************************************************************************/

	char* cmdinput = nullptr;

	for (int i = 1; i < argc; i += 2)
	{
		cmdinput = cmd_option_value(argv, argv + argc, std::string(argv[i]));

		if (argv[i] == std::string("-ip") || argv[i] == std::string("--infile_prefix"))
		{
			infile_prefix = cmdinput;
		}
		else if (argv[i] == std::string("-it") || argv[i] == std::string("--infile_type"))
		{
			infile_type = cmdinput;
			if (infile_type != ".bin" && infile_type != ".txt")
			{
				std::cerr << "Error. Invalid infile_type. infile_type must be .bin or .txt\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-hl") || argv[i] == std::string("--half_length"))
		{
			try
			{
				half_length = static_cast<dtype>(std::stod(cmdinput));
				if (half_length < std::numeric_limits<dtype>::min())
				{
					std::cerr << "Error. Invalid half_length input. half_length must be > " << std::numeric_limits<dtype>::min() << "\n";
					return -1;
				}
			}
			catch (...)
			{
				std::cerr << "Error. Invalid half_length input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-px") || argv[i] == std::string("--pixels"))
		{
			try
			{
				num_pixels = std::stoi(cmdinput);
				if (num_pixels < 1)
				{
					std::cerr << "Error. Invalid num_pixels input. num_pixels must be an integer > 0\n";
					return -1;
				}
			}
			catch (...)
			{
				std::cerr << "Error. Invalid num_pixels input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-os") || argv[i] == std::string("--over_sample"))
		{
			try
			{
				over_sample = std::stoi(cmdinput);
				if (over_sample < 0)
				{
					std::cerr << "Error. Invalid over_sample input. over_sample must be an integer >= 0\n";
					return -1;
				}
			}
			catch (...)
			{
				std::cerr << "Error. Invalid over_sample input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-wm") || argv[i] == std::string("--write_maps"))
		{
			try
			{
				write_maps = std::stoi(cmdinput);
				if (write_maps != 0 && write_maps != 1)
				{
					std::cerr << "Error. Invalid write_maps input. write_maps must be 1 (true) or 0 (false).\n";
					return -1;
				}
			}
			catch (...)
			{
				std::cerr << "Error. Invalid write_maps input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-wh") || argv[i] == std::string("--write_histograms"))
		{
			try
			{
				write_histograms = std::stoi(cmdinput);
				if (write_histograms != 0 && write_histograms != 1)
				{
					std::cerr << "Error. Invalid write_histograms input. write_histograms must be 1 (true) or 0 (false).\n";
					return -1;
				}
			}
			catch (...)
			{
				std::cerr << "Error. Invalid write_histograms input.\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-ot") || argv[i] == std::string("--outfile_type"))
		{
			outfile_type = cmdinput;
			if (outfile_type != ".bin" && outfile_type != ".txt")
			{
				std::cerr << "Error. Invalid outfile_type. outfile_type must be .bin or .txt\n";
				return -1;
			}
		}
		else if (argv[i] == std::string("-o") || argv[i] == std::string("--outfile_prefix"))
		{
			outfile_prefix = cmdinput;
		}
	}

	/****************************************************************************
	END read in options and values, checking correctness and exiting if necessary
	****************************************************************************/


	/*check that a CUDA capable device is present*/
	hipSetDevice(0);
	if (cuda_error("hipSetDevice", false, __FILE__, __LINE__)) return -1;


	std::string fname;

	/*read in parameter info and store necessary values*/
	if (infile_type == ".bin")
	{
		fname = infile_prefix + caustics_file + infile_type;
		std::cout << "Calculating some parameter values from caustics file " << fname << "\n";

		if (!read_params<int>(num_rows, num_cols, fname))
		{
			std::cerr << "Error. Unable to read parameter values from file " << fname << "\n";
			return -1;
		}

		std::cout << "Done calculating some parameter values from caustics file " << fname << "\n";
	}
	else
	{
		fname = infile_prefix + caustics_parameter_file;
		std::cout << "Calculating some parameter values from parameter info file " << fname << "\n";

		if (!read_params<int>(num_rows, num_cols, fname))
		{
			std::cerr << "Error. Unable to read parameter values from file " << fname << "\n";
			return -1;
		}

		std::cout << "Done calculating some parameter values from parameter info file " << fname << "\n";
	}
	std::cout << "\n";

	/*increase the number of pixels by 2^over_sample for initial sampling*/
	num_pixels <<= over_sample;


	/**********************
	BEGIN memory allocation
	**********************/

	std::cout << "Beginning memory allocation...\n";

	dtype* xpos = nullptr;
	dtype* ypos = nullptr;
	Complex<dtype>* caustics = nullptr;
	int* num_crossings = nullptr;

	hipMallocManaged(&xpos, num_rows * num_cols * sizeof(dtype));
	if (cuda_error("hipMallocManaged(*xpos)", false, __FILE__, __LINE__)) return -1;

	hipMallocManaged(&ypos, num_rows* num_cols * sizeof(dtype));
	if (cuda_error("hipMallocManaged(*ypos)", false, __FILE__, __LINE__)) return -1;

	hipMallocManaged(&caustics, num_rows* num_cols * sizeof(Complex<dtype>));
	if (cuda_error("hipMallocManaged(*caustics)", false, __FILE__, __LINE__)) return -1;

	hipMallocManaged(&num_crossings, num_pixels * num_pixels * sizeof(int));
	if (cuda_error("hipMallocManaged(*num_crossings)", false, __FILE__, __LINE__)) return -1;

	std::cout << "Done allocating memory.\n\n";

	/********************
	END memory allocation
	********************/


	/*variables for kernel threads and blocks*/
	dim3 threads;
	dim3 blocks;

	/*number of threads per block, and number of blocks per grid
	uses empirical values for maximum number of threads and blocks*/
	set_threads(threads, 16, 16);
	set_blocks(threads, blocks, num_rows, num_cols);


	if (infile_type == ".bin")
	{
		fname = infile_prefix + caustics_file + infile_type;
		std::cout << "Reading caustic positions from file " << fname << "\n";

		if (!read_complex_array<dtype>(caustics, num_rows, num_cols, fname))
		{
			std::cerr << "Error. Unable to read caustic positions from file " << fname << "\n";
			return -1;
		}

		std::cout << "Done reading caustic positions from file " << fname << "\n";
	}
	else
	{
		fname = infile_prefix + caustics_file + "_x" + infile_type;
		std::cout << "Reading caustic x positions from file " << fname << "\n";
		if (!read_re_array<dtype>(xpos, num_rows, num_cols, infile_prefix + caustics_file + "_x" + infile_type))
		{
			std::cerr << "Error. Unable to read caustic x positions from file " << fname << "\n";
			return -1;
		}
		std::cout << "Done reading caustic x positions from file " << fname << "\n";

		fname = infile_prefix + caustics_file + "_y" + infile_type;
		std::cout << "Reading caustic y positions from file " << fname << "\n";
		if (!read_re_array<dtype>(ypos, num_rows, num_cols, fname))
		{
			std::cerr << "Error. Unable to read caustic y positions from file " << fname << "\n";
			return -1;
		}
		std::cout << "Done reading caustic y positions from file " << fname << "\n";

		/*copy values into caustic array*/
		std::cout << "Copying caustic x and y positions into complex array...\n";
		copy_caustics_kernel<dtype> <<<blocks, threads>>> (xpos, ypos, caustics, num_rows, num_cols);
		if (cuda_error("copy_caustics_kernel", true, __FILE__, __LINE__)) return -1;
		std::cout << "Done copying caustic x and y positions into complex array.\n";
	}
	std::cout << "\n";


	/*redefine thread and block size to maximize parallelization*/
	set_threads(threads, 16, 16);
	set_blocks(threads, blocks, num_pixels, num_pixels);

	/*initialize pixel values*/
	initialize_pixels_kernel<dtype> <<<blocks, threads>>> (num_crossings, num_pixels);
	if (cuda_error("initialize_pixels_kernel", true, __FILE__, __LINE__)) return -1;


	/*redefine thread and block size to maximize parallelization*/
	set_threads(threads, 16, 16);
	set_blocks(threads, blocks, num_rows, num_cols);

	/*start and end time for timing purposes*/
	std::chrono::high_resolution_clock::time_point starttime;
	std::chrono::high_resolution_clock::time_point endtime;

	std::cout << "\nCalculating number of caustic crossings...\n";
	/*get current time at start of loop*/
	starttime = std::chrono::high_resolution_clock::now();
	find_num_caustic_crossings_kernel<dtype> <<<blocks, threads>>> (caustics, num_rows, num_cols, half_length, num_crossings, num_pixels);
	if (cuda_error("find_num_caustic_crossings_kernel", true, __FILE__, __LINE__)) return -1;
	/*get current time at end of loop, and calculate duration in milliseconds*/
	endtime = std::chrono::high_resolution_clock::now();
	double t_ncc = std::chrono::duration_cast<std::chrono::milliseconds>(endtime - starttime).count() / 1000.0;
	std::cout << "Done finding number of caustic crossings. Elapsed time: " << t_ncc << " seconds.\n\n";


	std::cout << "Downsampling number of caustic crossings...\n";
	/*get current time at start of loop*/
	starttime = std::chrono::high_resolution_clock::now();

	for (int i = 0; i < over_sample; i++)
	{
		num_pixels >>= 1;

		/*redefine thread and block size to maximize parallelization*/
		set_threads(threads, 16, 16);
		set_blocks(threads, blocks, num_pixels, num_pixels);

		reduce_pix_array_kernel<dtype> <<<blocks, threads>>> (num_crossings, num_pixels);
		if (cuda_error("reduce_pix_array_kernel", true, __FILE__, __LINE__)) return -1;

		/*redefine thread and block size to maximize parallelization*/
		set_threads(threads, 512);
		set_blocks(threads, blocks, num_pixels);

		for (int j = 1; j < num_pixels; j++)
		{
			shift_pix_column_kernel<dtype> <<<blocks, threads>>> (num_crossings, num_pixels, j);
		}
		for (int j = 1; j < num_pixels; j++)
		{
			shift_pix_row_kernel<dtype> <<<blocks, threads>>> (num_crossings, num_pixels, j);
		}
		if (cuda_error("shift_pix_kernel", true, __FILE__, __LINE__)) return -1;
	}

	/*get current time at end of loop, and calculate duration in milliseconds*/
	endtime = std::chrono::high_resolution_clock::now();
	double t_reduce = std::chrono::duration_cast<std::chrono::milliseconds>(endtime - starttime).count() / 1000.0;
	std::cout << "Done downsampling number of caustic crossings. Elapsed time: " << t_reduce << " seconds.\n\n";


	/********************************
	create histograms of pixel values
	********************************/

	int* min_num = nullptr;
	int* max_num = nullptr;

	int* histogram = nullptr;

	int histogram_length = 0;

	if (write_histograms)
	{
		std::cout << "Creating histograms...\n";

		hipMallocManaged(&min_num, sizeof(int));
		if (cuda_error("hipMallocManaged(*min_rays)", false, __FILE__, __LINE__)) return -1;
		hipMallocManaged(&max_num, sizeof(int));
		if (cuda_error("hipMallocManaged(*max_rays)", false, __FILE__, __LINE__)) return -1;

		*min_num = std::numeric_limits<int>::max();
		*max_num = std::numeric_limits<int>::min();

		/*redefine thread and block size to maximize parallelization*/
		set_threads(threads, 16, 16);
		set_blocks(threads, blocks, num_pixels, num_pixels);

		histogram_min_max_kernel<dtype> <<<blocks, threads>>> (num_crossings, num_pixels, min_num, max_num);
		if (cuda_error("histogram_min_max_kernel", true, __FILE__, __LINE__)) return -1;

		histogram_length = *max_num - *min_num + 1;

		hipMallocManaged(&histogram, histogram_length * sizeof(int));
		if (cuda_error("hipMallocManaged(*histogram)", false, __FILE__, __LINE__)) return -1;

		/*redefine thread and block size to maximize parallelization*/
		set_threads(threads, 512);
		set_blocks(threads, blocks, histogram_length);

		initialize_histogram_kernel<dtype> <<<blocks, threads>>> (histogram, histogram_length);
		if (cuda_error("initialize_histogram_kernel", true, __FILE__, __LINE__)) return -1;

		/*redefine thread and block size to maximize parallelization*/
		set_threads(threads, 16, 16);
		set_blocks(threads, blocks, num_pixels, num_pixels);

		histogram_kernel<dtype> <<<blocks, threads>>> (num_crossings, num_pixels, *min_num, histogram);
		if (cuda_error("histogram_kernel", true, __FILE__, __LINE__)) return -1;

		std::cout << "Done creating histograms.\n\n";
	}
	/***************************************
	done creating histograms of pixel values
	***************************************/


	/*stream for writing output files
	set precision to 9 digits*/
	std::ofstream outfile;
	outfile.precision(9);

	std::cout << "Writing parameter info...\n";
	fname = outfile_prefix + "ncc_parameter_info.txt";
	outfile.open(fname);
	if (!outfile.is_open())
	{
		std::cerr << "Error. Failed to open file " << fname << "\n";
		return -1;
	}
	outfile << "half_length " << half_length << "\n";
	outfile << "num_pixels " << num_pixels << "\n";
	outfile << "over_sample " << over_sample << "\n";
	outfile << "t_ncc " << t_ncc << "\n";
	outfile.close();
	std::cout << "Done writing parameter info to file " << fname << "\n\n";


	/*histogram of number of caustic crossings map*/
	if (write_histograms)
	{
		std::cout << "Writing number of caustic crossings histogram...\n";
		fname = outfile_prefix + "ncc_ncc_numpixels.txt";
		if (!write_histogram<dtype>(histogram, histogram_length, *min_num, fname))
		{
			std::cerr << "Error. Unable to write caustic crossings histogram to file " << fname << "\n";
			return -1;
		}
		std::cout << "Done writing number of caustic crossings histogram to file " << fname << "\n\n";
	}
	

	if (write_maps)
	{
		/*write number of caustic crossings*/
		std::cout << "Writing number of caustic crossings...\n";
		fname = outfile_prefix + "ncc_ncc" + outfile_type;
		if (!write_array<int>(num_crossings, num_pixels, num_pixels, fname))
		{
			std::cerr << "Error. Unable to write number of caustic crossings to file " << fname << "\n";
			return -1;
		}
		std::cout << "Done writing number of caustic crossings to file " << fname << "\n\n";
	}

	std::cout << "Done.\n";

	hipDeviceReset();
	if (cuda_error("hipDeviceReset", false, __FILE__, __LINE__)) return -1;

	return 0;
}

